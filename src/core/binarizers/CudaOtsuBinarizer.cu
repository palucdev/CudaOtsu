#include "hip/hip_runtime.h"
#include "CudaOtsuBinarizer.cuh"

#include <stdio.h>

// CUDA imports
#include <hip/hip_runtime.h>

__global__ void kernelCalculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			int pixelValue = (int)rawPixels[i];
			atomicAdd(&histogram[pixelValue], 1);
		}
	}
}

__global__ void kernelComputeClassVariances(double* histogram, double allProbabilitySum, long int totalPixels, double* betweenClassVariance)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	double firstClassProbability = 0, secondClassProbability = 0;
	double firstClassMean = 0, secondClassMean = 0;
	double firstProbabilitySum = 0;

	for (int t = 0; t <= id % PngImage::MAX_PIXEL_VALUE; t++) {
		firstClassProbability += histogram[t];
		firstProbabilitySum += t * histogram[t];
	}

	secondClassProbability = 1 - firstClassProbability;

	firstClassMean = (double)firstProbabilitySum / (double)firstClassProbability;
	secondClassMean = (double)(allProbabilitySum - firstProbabilitySum) / (double)secondClassProbability;

	betweenClassVariance[id] = firstClassProbability * secondClassProbability * pow((firstClassMean - secondClassMean), 2);
}

__global__ void kernelBinarize(unsigned char* rawPixels, long totalPixels, long chunkSize, unsigned char threshold)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			if ((int)rawPixels[i] >(int)threshold) {
				rawPixels[i] = PngImage::COLOR_WHITE;
			}
			else {
				rawPixels[i] = PngImage::COLOR_BLACK;
			}
		}
	}
}

CudaOtsuBinarizer::CudaOtsuBinarizer(int threadsPerBlock, int numBlocks, bool drawHistogram, const char* TAG) {
	this->threadsPerBlock_ = threadsPerBlock;
	this->numBlocks_ = numBlocks;
	this->binarizerTimestamp_ = new ExecutionTimestamp();

	this->drawHistogram_ = drawHistogram;
	this->TAG = TAG;
}

CudaOtsuBinarizer::~CudaOtsuBinarizer() {
	delete this->binarizerTimestamp_;
}

PngImage* CudaOtsuBinarizer::binarize(PngImage * imageToBinarize)
{
	long totalImagePixels = (long)imageToBinarize->getRawPixelData().size();

	double* histogram = cudaCalculateHistogram(imageToBinarize->getRawPixelData().data(), totalImagePixels);
	hipDeviceSynchronize();
	
	if (this->drawHistogram_) {
		showHistogram(histogram);
	}

	unsigned char threshold;
	threshold = cudaFindThreshold(histogram, totalImagePixels);
	hipDeviceSynchronize();
	printf("\t[%s] Threshold value: %d\n", this->TAG, threshold);

	delete histogram;

	unsigned char* binarizedRawPixels = cudaBinarize(imageToBinarize->getRawPixelData().data(), totalImagePixels, threshold);
	hipDeviceSynchronize();

	std::vector<unsigned char> binarizedVector(&binarizedRawPixels[0], &binarizedRawPixels[totalImagePixels]);

	delete binarizedRawPixels;

	printf("\n\t[%s] Total calculation time: %.6f milliseconds \n", this->TAG, binarizerTimestamp_->getExecutionTime());

	return new PngImage(
		imageToBinarize->getFilename(),
		imageToBinarize->getWidth(),
		imageToBinarize->getHeight(),
		binarizedVector
	);
}

std::string CudaOtsuBinarizer::getBinarizerExecutionInfo(std::string fileName)
{
	return binarizerTimestamp_->toCommaSeparatedRow(fileName, std::string(this->TAG));
}

void CudaOtsuBinarizer::showHistogram(double* histogram) {
	printf("\nHistogram:\n");
	double value = 0;
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		value = histogram[i];
		printf("\tPixel value %d -> %.5f\n", i, value);
	}
}

double* CudaOtsuBinarizer::cudaCalculateHistogram(unsigned char* rawPixels, long totalPixels) {
	//TODO: check hipGetDeviceProperties function!

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned int* hostHistogram = new unsigned int[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostHistogram[i] = 0;
	}

	unsigned int* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, hostHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock_ * numBlocks_)) + 1;

	hipEventRecord(start);
	kernelCalculateHistogram<<<numBlocks_, threadsPerBlock_>>>(deviceHistogram, deviceRawPixels, chunkSize, totalPixels);
	hipEventRecord(stop);

	hipMemcpy(hostHistogram, deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Histogram calculated in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->histogramBuildingTimeInSeconds += milliseconds * 1000;

	hipFree(deviceHistogram);
	hipFree(deviceRawPixels);

	double* normalizedHistogram = new double[PngImage::MAX_PIXEL_VALUE];
	long pixelsSum = 0;
	for (int v = 0; v < PngImage::MAX_PIXEL_VALUE; v++) {
		normalizedHistogram[v] = (double)hostHistogram[v] / (double)totalPixels;
		pixelsSum += hostHistogram[v];
	}
	printf("\n\t[%s] Histogram pixels: %d \n", this->TAG, pixelsSum);

	delete hostHistogram;

	return normalizedHistogram;
}

unsigned char CudaOtsuBinarizer::cudaFindThreshold(double* histogram, long int totalPixels) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int threadsPerBlock = 256;
	int numBlocks = 1;

	double allProbabilitySum = 0;
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		allProbabilitySum += i * histogram[i];
	}

	double* hostBetweenClassVariances = new double[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostBetweenClassVariances[i] = 0;
	}

	double* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(double) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, histogram, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	double* deviceBetweenClassVariances;
	hipMalloc((void **)&deviceBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceBetweenClassVariances, hostBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	hipEventRecord(start);
	kernelComputeClassVariances<<<numBlocks, threadsPerBlock>>>(deviceHistogram, allProbabilitySum, totalPixels, deviceBetweenClassVariances);
	hipEventRecord(stop);
	hipMemcpy(hostBetweenClassVariances, deviceBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Threshold calculated in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->thresholdFindingTimeInSeconds += milliseconds * 1000;

	hipFree(deviceHistogram);
	hipFree(deviceBetweenClassVariances);

	double maxVariance = 0;
	unsigned char currentBestThreshold = 0;
	for (int t = 0; t < PngImage::MAX_PIXEL_VALUE; t++) {
		if (hostBetweenClassVariances[t] > maxVariance) {
			currentBestThreshold = (unsigned char)t;
			maxVariance = hostBetweenClassVariances[t];
		}
	}

	delete hostBetweenClassVariances;

	return currentBestThreshold;
}

unsigned char* CudaOtsuBinarizer::cudaBinarize(unsigned char * rawPixels, long totalPixels, unsigned char threshold) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned char* hostRawPixels = new unsigned char[totalPixels];

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, totalPixels * sizeof(unsigned char), hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock_ * numBlocks_)) + 1;

	hipEventRecord(start);
	kernelBinarize<<<numBlocks_, threadsPerBlock_>>>(deviceRawPixels, totalPixels, chunkSize, threshold);
	hipEventRecord(stop);

	hipMemcpy(hostRawPixels, deviceRawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\t[%s] Binarized in %.6f milliseconds \n", this->TAG, milliseconds);
	binarizerTimestamp_->binarizationTimeInSeconds += milliseconds * 1000;

	hipFree(deviceRawPixels);

	return hostRawPixels;
}
